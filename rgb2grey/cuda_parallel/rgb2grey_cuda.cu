// CUDA multiple threads

#include <hip/hip_runtime.h>


__global__
void rgb2grey_kernel(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
    int idx = threadIdx.x+ blockIdx.x* blockDim.x;

    if (idx < numCols*numRows) {
        uchar4 rgba = rgbaImage[idx];
        float grey = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
        greyImage[idx] = grey;
    }
}

void rgb2grey_cuda( const uchar4* const h_rgbaImage,
                        unsigned char *const h_greyImage,
                        size_t numRows, size_t numCols)
{
    uchar4        *d_rgbaImage;
    unsigned char *d_greyImage;
    size_t numPixels= numRows*numCols;

    // Alloc memory
    hipMalloc((void **) &d_rgbaImage,  sizeof(uchar4) * numPixels);
    hipMalloc((void **) &d_greyImage,  sizeof(uchar4) * numPixels);

    // Copy from Host to Device
    hipMemset(d_greyImage, 0, numPixels * sizeof(unsigned char));
    hipMemcpy(d_rgbaImage, h_rgbaImage, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice);

    // Kernel Launch
    const int threadPerBlock = 512;
    const int numBlock = numPixels/threadPerBlock+1;

    rgb2grey_kernel<<<numBlock, threadPerBlock>>>(d_rgbaImage, d_greyImage, numRows, numCols);

    // Copy from Device to Host
    hipMemcpy(h_greyImage, d_greyImage, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost);

    // Free memory
    hipDeviceSynchronize();
    hipFree(d_greyImage);
    hipFree(d_rgbaImage);
}
