#include "hip/hip_runtime.h"
// Share Memory

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "timer.h"

__global__
void rgb_to_grey_global(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
    int idx = threadIdx.x+ blockIdx.x* blockDim.x;
    
    if (idx < numCols*numRows) {
        uchar4 rgba = rgbaImage[idx];
        float grey = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
        greyImage[idx] = grey;
    }
}

__global__
void rgb_to_grey_shared(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
    int idx = threadIdx.x+ blockIdx.x* blockDim.x;
    
    __shared__ uchar4 sh_rgba[256];
    sh_rgba[threadIdx.x] = rgbaImage[idx];
    
    // barrier
    
    
    // __shared__ uchar4 grey[256];
    
    if (idx < numCols*numRows) {
        uchar4 rgba = sh_rgba[threadIdx.x];
        float grey = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
        greyImage[idx] = grey;
    }
}

void rgba_to_grey_cuda( const uchar4* const h_rgbaImage,
                        unsigned char *const h_greyImage,
                        size_t numRows, size_t numCols)
{
    uchar4        *d_rgbaImage;
    unsigned char *d_greyImage;
    size_t numPixels= numRows*numCols;
        
    // Alloc memory
    hipMalloc((void **) &d_rgbaImage,  sizeof(uchar4) * numPixels);
    hipMalloc((void **) &d_greyImage,  sizeof(uchar4) * numPixels);
    
    // Copy from Host to Device
    hipMemset(d_greyImage, 0, numPixels * sizeof(unsigned char));
    hipMemcpy(d_rgbaImage, h_rgbaImage, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice);
    
    // Kernel Lunch
    const int threadPerBlock = 256;
    const int numBlock = numPixels/threadPerBlock+1;
    
    GpuTimer timer1;
    timer1.Start();
    rgb_to_grey_global<<<numBlock, threadPerBlock>>>(d_rgbaImage, d_greyImage, numRows, numCols);
    timer1.Stop();
    std::cout << "On global memory ran in: " << timer1.Elapsed() << "msecs." << std::endl;
    
/*     GpuTimer timer2;
    timer2.Start();
    rgb_to_grey_shared<<<numBlock, threadPerBlock>>>(d_rgbaImage, d_greyImage, numRows, numCols);
    timer2.Stop();
    std::cout << "On shared memory ran in: " << timer2.Elapsed() << "msecs." << std::endl; */
    
    // Copy from Device to Host
    hipMemcpy(h_greyImage, d_greyImage, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost);
        
    // Free memory
    hipDeviceSynchronize(); 
    hipFree(d_greyImage);
    hipFree(d_rgbaImage);
}

